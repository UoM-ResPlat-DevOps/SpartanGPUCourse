
#include <hip/hip_runtime.h>
#include <stdio.h> 
#define N (16*16)
#define M (20)

  __global__ void cudakernel(float *buf)
  {
     int i = threadIdx.x + blockIdx.x * blockDim.x;
     buf[i] = 1.0f * i / N;
     for(int j = 0; j < M; j++)
        buf[i] = buf[i] * buf[i] - 0.25f;
  }

  int main()
  {
     float data[N];
     float *d_data;
     hipMalloc(&d_data, N * sizeof(float));
     cudakernel<<<N/32, 32>>>(d_data);
     hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
     hipFree(d_data); 

     int sel;
     printf("Enter an index: ");
     scanf("%d", &sel);
     printf("data[%d] = %f\n", sel, data[sel]);
  }

