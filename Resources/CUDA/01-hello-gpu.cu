
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 */

void helloGPU()
{
  printf("Hello also from the CPU.\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   */

  helloGPU();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
}
