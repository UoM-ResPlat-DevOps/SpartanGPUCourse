
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  loop(N);
}