#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)

{
	if (result != hipSuccess) {
	fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
	assert(result == hipSuccess);
	}
	return result;
}

int main()
{

/*
* The macro can be wrapped around any function returning
* a value of type `hipError_t`.
*/

	checkCuda( hipDeviceSynchronize() )
}
