
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define checkCudaError(status) { \
	if(status != hipSuccess) { \
		std::cout << "CUDA Error " << __FILE__ << ", " << __LINE__ \
			<< ": " << hipGetErrorString(status) << "\n"; \
		exit(-1); \
	} \
}

__global__ void vecAdd(int * a, int * b, int * c, int size) {

	//ADD CODE HERE
	int i = threadIdx.x;
	int j = blockIdx.x*blockDim.x;
	printf("I am in: %d, %d\n", i , j);
	c[i + j] = a[i + j] + b[i + j];
}

int main() {

	//checkCudaError(cudaSetDevice(1));
	int device;
	checkCudaError(hipGetDevice(&device));
	hipDeviceProp_t prop;
	checkCudaError(hipGetDeviceProperties(&prop, device));
	std::cout << "Device " << device << ": " << prop.name << "\n";
	std::cout << "GPU Cores: " << prop.multiProcessorCount << "\n";
	std::cout << "Compute Capability: " << prop.major << "." << prop.minor << "\n";

	const int GRID_SIZE = 16;
	const int CTA_SIZE = 128;
	const int size = GRID_SIZE * CTA_SIZE;
	int * a, * b, * c;
	int * dev_a, * dev_b, * dev_c;

	a = (int *) malloc (sizeof(int) * size);
	b = (int *) malloc (sizeof(int) * size);
	c = (int *) malloc (sizeof(int) * size);
	if(!a || !b || !c) {
		std::cout << "Error: out of memory\n";
		exit(-1);
	}

	for(int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = i+1;
	}
	memset(c, 0, sizeof(int) * size);

	checkCudaError(hipMalloc(&dev_a, sizeof(int) * size));
	checkCudaError(hipMalloc(&dev_b, sizeof(int) * size));	
	checkCudaError(hipMalloc(&dev_c, sizeof(int) * size));	
	
	checkCudaError(hipMemcpy(dev_a, a, sizeof(int) * size, hipMemcpyHostToDevice));
	checkCudaError(hipMemcpy(dev_b, b, sizeof(int) * size, hipMemcpyHostToDevice));
	checkCudaError(hipMemset(dev_c, 0, sizeof(int) * size));

	vecAdd<<<GRID_SIZE, CTA_SIZE>>>(dev_a, dev_b, dev_c, size);

	checkCudaError(hipDeviceSynchronize());
	checkCudaError(hipMemcpy(c, dev_c, sizeof(int) * size, hipMemcpyDeviceToHost));

	for(int i = 0; i < size; i++) {
//		std::cout << i << ": " << c[i] << "\n";
		if(c[i] != i*2+1) {
			std::cout << "Error: c[" << i << "] != " <<
				i*2+1 << "\n";
			exit(-1);
		}
	}
	std::cout << "Pass\n";
}