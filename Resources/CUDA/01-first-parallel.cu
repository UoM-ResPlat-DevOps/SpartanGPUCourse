
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor firstParallel so that it can run on the GPU.
 */

void firstParallel()
{
  printf("This should be running in parallel.\n");
}

int main()
{
  /*
   * Refactor this call to firstParallel to execute in parallel
   * on the GPU.
   */

  firstParallel();

  /*
   * Some code is needed below so that the CPU will wait
   * for the GPU kernels to complete before proceeding.
   */

}